#include "hip/hip_runtime.h"
#include "abschlussarbeit.cuh"

#define left left2d[tid]
#define oleft oleft2d[tid]

__global__ void computeDistanceMatrix(const unsigned int *prev, const unsigned int *last,
									  const unsigned short *other, const bool *isleaf,
									  float (*d)[NOV], float delta, 
									  bool (*left2d)[NOV], bool (*oleft2d)[NOV], unsigned char (*a)[NOV])
{
	unsigned short tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < NOV && !isleaf[tid]){
		for (unsigned short i = tid + 1; i < NOV; i++) if (!isleaf[i] && a[tid][i]<4) {
			for (unsigned int j = 0; j < NOV; j++)
				left[j] = oleft[j] = false;
			unsigned short num_left = 0;
			unsigned short num_right = 0;
			unsigned int ps = last[tid], pt = last[i];
			unsigned short t = 1;
			while (ps) {
				unsigned short otherps = other[ps];
				if (otherps != i){
					left[otherps] = oleft[otherps] = true;
					num_left++;
				}
				ps = prev[ps];
			}
			while (pt) {
				unsigned short otherpt = other[pt];
				if (otherpt == tid) {
					d[tid][i]+=1;
					pt = prev[pt];
					continue;
				}
				if (left[otherpt]) {
					d[tid][i]+=0.5;
					left[otherpt] = false;
					num_left--;
					pt = prev[pt];
					continue;
				}
				pt = prev[pt];
			}
			pt = last[i];
			while (pt) {
				bool isright = false;
				unsigned short otherpt = other[pt];
				if (!oleft[otherpt] && otherpt != tid){
					unsigned int p = last[otherpt];
					while (p) {
						unsigned short otherp = other[p];
						if (left[otherp]) {
							isright = true;
							left[otherp] = false;
							break;
						}
						p = prev[p];
					}
				}
				if (isright) {
					num_right++;
				}
				pt = prev[pt];
			}
			if (num_left != 0 && num_right != 0){
				unsigned int ans;
				if (num_left<num_right)
					ans = num_left;
				else
					ans = num_right;
				d[tid][i] += ans * 0.33333333333;				
			}
		}
		for (unsigned int i = tid+1; i < NOV; i++)
			d[i][tid] = d[tid][i];
		__syncthreads();
		for (unsigned int i = 0; i < NOV; i++)
			if (i != tid){
				if (d[tid][i]<0.00001 && d[tid][i]>-0.00001)
					d[tid][i] = 99999;
				else
					if (a[tid][i])
						d[tid][i] = (1/d[tid][i])*(1-delta);
					else
						d[tid][i] = 1/d[tid][i];
			}
	}
}

__global__ void matrixMulCUDA(unsigned char (*C)[NOV], 
							  unsigned char (*A)[NOV], 
							  unsigned char (*B)[NOV])
{
	unsigned int tid = blockDim.x * blockIdx.x +threadIdx.x;
	unsigned int cx = tid / NOV;
	unsigned int cy = tid % NOV;
	if (cx<NOV && cy<NOV){
		C[cx][cy] = 0;
#pragma unroll
		for (unsigned int i = 0; i < NOV; i++)
			C[cx][cy]|=A[cx][i]&B[i][cy];
	}
}

__global__ void matrixfix(unsigned char (*A), 
						  const unsigned char (*B),
						  const unsigned char (*C),
						  const unsigned char (*D))
{
	unsigned int tid = blockDim.x * blockIdx.x +threadIdx.x;
	unsigned int cx = tid / NOV;
	unsigned int cy = tid % NOV;
	if (cx >= NOV || cy >= NOV) return;
	if (cx == cy){
		A[tid] = 0;
		return;
	}
	if (B[tid]) {
		A[tid] = 1;
		return;
	}
	if (C[tid]) {
		A[tid] = 2;
		return;
	}
	if (D[tid]) {
		A[tid] = 3;
		return;
	}
	A[tid] = 99;
}

__global__ void Muld(unsigned char * A, unsigned char * B, int wA, int wB, unsigned char * C)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
 
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int aBegin = wA * BLOCK_SIZE * by;
	int aEnd = aBegin + wA - 1;
	int aStep = BLOCK_SIZE;
	int bBegin = BLOCK_SIZE * bx;
	int bStep = BLOCK_SIZE * wB;
	
	unsigned char Csub = 0;
	for (int a = aBegin, b = bBegin;
	a <= aEnd;
	a += aStep, b += bStep) {
		if (b + wB * ty + tx < NOV * NOV){
			__shared__ unsigned char As[BLOCK_SIZE][BLOCK_SIZE];
			__shared__ unsigned char Bs[BLOCK_SIZE][BLOCK_SIZE];
			As[ty][tx] = A[a + wA * ty + tx];
			Bs[ty][tx] = B[b + wB * ty + tx];
			__syncthreads();
			for (int k = 0; k < BLOCK_SIZE; ++k)
				Csub += As[ty][k] * Bs[k][tx];
			__syncthreads();
		}
	}
	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
} 