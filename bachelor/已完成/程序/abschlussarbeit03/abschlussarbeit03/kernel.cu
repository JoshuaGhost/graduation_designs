#include "hip/hip_runtime.h"
#include "abschlussarbeit.cuh"
#include "dis_mat.cuh"
#include "min_reduction.cuh"
#include "sep_mst.cuh"

unsigned int n = NOV;
unsigned int m;
unsigned int *prev;
unsigned int *last;
unsigned short *other;
unsigned int rank;

unsigned int *prev1, *prev2;
unsigned int *last1, *last2;
unsigned short *other1, *other2;
float *weight1, *weight2;
float dslen1 = 0.0, dslen2 = 0.0;

bool (*inlgroup)[NOV];
bool *left;

unsigned short epsilon = 3, theta = 3;
float delta = 0.3;

unsigned short *fa;
unsigned int num_leaves = 0;
bool *isleaf;

float (*d)[NOV];
unsigned char (*a)[NOV];


class myException:public std::exception  
{  
public:  
    myException():exception("ERROR, Can't open file!\n")  
    {  
    }  
};

void readfile_edgelist()
{
	FILE *fp;
	try{
		fp = fopen("e:\\data_200x200x3820.txt","r");
		if (!fp)
			throw myException();
	}
	catch (myException& me){
		std::cout << me.what();
		system("pause");
		return;
	}
	fscanf(fp, "%d %d %f", &n, &m, &delta);
	for (unsigned int i = 1; i <= m; i++){
		unsigned short x, y, z;
		fscanf(fp,"%u %u %u", &x, &y, &z);
		x--;y--;
		prev[i] = last[x];
		last[x] = i;
		other[i] = y;
		prev[i + NOE] = last[y];
		last[y] = NOE + i;
		other[NOE + i] = x;
		a[x][y] = a[y][x] = 1;
	}
	fclose(fp);
}

void readfile_matrix()
{
	FILE *fp;
	try{
		fp = fopen("e:\\data_50x50x808.txt","r");
		if (!fp)
			throw myException();
	}
	catch (myException& me){
		std::cout << me.what();
		system("pause");
		return;
	}

	m = 0;
	int x;
	for (unsigned short i = 0; i < NOV; i++){
		a[i][i] = 0;
		for (unsigned short j = 0; j < NOV; j++){
			fscanf(fp, "%d", &x);
			if (x){
				m++;
				prev[m] = last[i];
				last[i] = m;
				other[m] = j;
				a[i][j] = (short)x;
			}
		}
	}
	fclose(fp);
}

void cudaMatrixMul()
{
	unsigned int size = NOV * NOV;
	unsigned int mem_size = sizeof(unsigned char) * size;
	unsigned char (*d_A)[NOV],(*d_B)[NOV],(*d_C)[NOV],(*d_D)[NOV];
    hipMalloc((void **) &d_A, mem_size);
	hipMalloc((void **) &d_B, mem_size);
	hipMalloc((void **) &d_C, mem_size);
	hipMalloc((void **) &d_D, mem_size);
	hipMemcpy(d_A, a, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, d_A, mem_size, hipMemcpyDeviceToDevice);

	unsigned int num_threads = (NOV*NOV)>1024 ? 1024 : NOV;
	unsigned int num_blocks = ((NOV*NOV)/num_threads) + (((NOV*NOV)%num_threads) ? 1 : 0);
	matrixMulCUDA<<<num_blocks, num_threads>>>(d_C, d_A, d_B);
	hipDeviceSynchronize();
	matrixMulCUDA<<<num_blocks, num_threads>>>(d_D, d_C, d_B);
	hipDeviceSynchronize();
	matrixfix<<<num_blocks, num_threads>>>(d_A, d_B, d_C, d_D);
	hipDeviceSynchronize();
	hipMemcpy(a, d_A, mem_size, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_D);
}

unsigned short find(unsigned short p)
{
	if (fa[p] == p) 
		return p;
	fa[p] = find(fa[p]);
	return fa[p];
}

void search_leaves()
{
	unsigned short q[NOV];
	unsigned short head = 0, tail = 0;

	for (unsigned int i = 0; i < NOV; i++)
		fa[i] = i;

	for (int i = 0; i < NOV; i++)
		if (!prev[last[i]]) {
			q[head] = i;
			head++;
		}
	while (head > tail) {
		unsigned short num = 0;
		unsigned short father;
		unsigned int p = last[q[tail]];
		while (p){
			if (!isleaf[other[p]]){
				num++;
				father = other[p];
			}
			do {
			p = prev[p];
			}while (0);
		}
		if (num == 1){
			isleaf[q[tail]] = true;
			num_leaves++;
			fa[q[tail]] = find(father);
		}
		tail++;
	}
}

void cudaComputeDistanceMatrix()
{
	unsigned int *d_prev;
	unsigned int *d_last;
	unsigned short *d_other;
	bool *d_isleaf;
	float (*d_d)[NOV];
	bool (*d_left2d)[NOV];
	bool (*d_right2d)[NOV];
	unsigned char (*d_a)[NOV];

	hipMalloc((void **)&d_prev, (2 * NOE + 2)*sizeof(unsigned int));
	hipMalloc((void **)&d_last, NOV*sizeof(int));
	hipMalloc((void **)&d_other, (2 * NOE + 2)*sizeof(unsigned short));
	hipMalloc((void **)&d_isleaf, NOV*sizeof(bool));
	hipMalloc((void **)&d_d, NOV*NOV*sizeof(float));
	hipMalloc((void **)&d_right2d, NOV*NOV*sizeof(bool));
	hipMalloc((void **)&d_left2d, NOV*NOV*sizeof(bool));
	hipMalloc((void **)&d_a, NOV*NOV*sizeof(unsigned char));

	hipMemcpy(d_d, d, NOV*NOV*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_prev, prev, (2 * NOE + 2)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_last, last, NOV*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_other, other, (2 * NOE + 2)*sizeof(unsigned short), hipMemcpyHostToDevice);
	hipMemcpy(d_isleaf, isleaf, NOV*sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(d_a, a, NOV*NOV*sizeof(unsigned char), hipMemcpyHostToDevice);

	int num_threads = 1024, num_blocks = NOV / num_threads + (NOV % num_threads ? 1 : 0);
	computeDistanceMatrix<<<num_blocks, num_threads>>>(d_prev, d_last, d_other, d_isleaf, d_d, delta, 
													   d_left2d, d_right2d, d_a);
	hipMemcpy(d, d_d, NOV*NOV*sizeof(float), hipMemcpyDeviceToHost);
	printf("%d\n",hipGetLastError());
	hipFree(d_prev);
	hipFree(d_last);
	hipFree(d_other);
	hipFree(d_d);
}

void prim_mst(const unsigned int *v, const unsigned short *e, float *w, const unsigned int times)
{
	float *d_iweight = NULL, *d_oweight = NULL;
	unsigned int *d_oid = NULL;
	int num_threads = 1024;
	int num_blocks = NOV / num_threads+(NOV % num_threads == 0?0:1);
	hipMalloc((void **)&d_iweight, n * sizeof(float));
	hipMalloc((void **)&d_oid, sizeof(int));
	hipMalloc((void **)&d_oweight, sizeof(float));

	unsigned int i, j;
	unsigned int ki;
	unsigned short k;
	unsigned int p = 1;
	unsigned int closest[NOV];
	float value[NOV] = {0};
	for (i = 1; i<n; i++)
		if (isleaf[i]) 
			value[i] = 0;
		else
			value[i] = MAX_WEIGHT;
	for (i = 0; i < NOV; i++)
		if (!isleaf[i])
			break;
	for (j = v[i]; j < v[i+1]; j++)
		value[e[j]] = w[j];
	value[i] = 0;
	for (j = 0; j < NOV; j++)
		closest[j] = i;

	for (i = 0; i < n-1-num_leaves; i++){
		
		hipMemcpy(d_iweight, value, n * sizeof(float), hipMemcpyHostToDevice);
		if (n == 5000){
			unsigned int resid[5];
			float resweight[5];
			for (unsigned j = 0; j < 5; j++)
				if (j < 4){
					hipMemcpy(d_iweight, &value[j*1024], 1024*sizeof(float), hipMemcpyHostToDevice);
					min_reduction1<<<1,num_threads>>>(d_iweight, d_oweight, d_oid, 1024);
					hipMemcpy(&resweight[j], d_oweight, sizeof(float), hipMemcpyDeviceToHost);
					hipMemcpy(&resid[j], d_oid, sizeof(int), hipMemcpyDeviceToHost);
				} else {
					hipMemcpy(d_iweight, &value[4096], 904*sizeof(float), hipMemcpyHostToDevice);
					min_reduction1<<<1, num_threads>>>(d_iweight, d_oweight, d_oid, 904);
					hipMemcpy(&resweight[j], d_oweight, sizeof(float), hipMemcpyDeviceToDevice);
					hipMemcpy(&resid[j], d_oid, sizeof(int), hipMemcpyDeviceToHost);
				}
			unsigned int *d_iid = NULL;
			hipMalloc((void **)&d_iid, 5*sizeof(int));
			hipMemcpy(d_iid, resid, 5*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(d_iweight, resweight, 5*sizeof(float), hipMemcpyHostToDevice);
			min_reduction2<<<1, 5>>>(d_iweight, d_oweight, d_iid, d_oid, 5);
		} else {
			min_reduction1<<<num_blocks, num_threads>>>(d_iweight, d_oweight, d_oid, n);
		}
		hipMemcpy(&ki, d_oid, sizeof(int), hipMemcpyDeviceToHost);
		k = (unsigned short)ki;
		if (times == 1){
			prev1[p] = last1[closest[k]];
			last1[closest[k]] = p;
			other1[p] = k;
			prev1[p + NOV - 1] = last1[k];
			last1[k] = p + NOV - 1;
			other1[p + NOV - 1] = closest[k];
			weight1[p] = value[k];
			weight1[p + NOV - 1] = weight1[p];
			dslen1 += value[k];
			for (j = v[closest[k]]; j< v[closest[k]+1]; j++){
				if (e[j] == k){
					w[j] = MAX_WEIGHT;
					break;
				}
			}
			for (j = v[k]; j < v[k+1]; j++){
				if (e[j] == closest[k]){
					w[j] = MAX_WEIGHT;
					break;
				}
			}
		}else{
			prev2[p] = last2[closest[k]];
			last2[closest[k]] = p;
			other2[p] = k;
			prev2[p + NOV - 1] = last2[k];
			last2[k] = p + NOV - 1;
			other2[p + NOV - 1] = closest[k];
			weight2[p] = value[k];
			weight2[p + NOV - 1] = weight2[p];
			dslen2 += value[k];
		}
		p++;
		value[k] = 0;
		for (j = v[k]; j < v[k+1]; j++)
			if (w[j] < value[e[j]]) {
				value[e[j]] = w[j];
				closest[e[j]] = k;
			}
	}
	hipFree(d_iweight);
	hipFree(d_oweight);
	hipFree(d_oid);
	p = NOV - num_leaves;
	if (times == 1) {
		for (unsigned short i = 0; i < NOV; i++){
			if (fa[i] != i) {
				prev1[p] = last1[i];
				last1[i] = p;
				other1[p] = fa[i];
				prev1[p+NOV-1] = last1[fa[i]];
				last1[fa[i]] = p+NOV-1;
				other1[p+NOV-1] = i;
			}
		}
	}else{
		for (unsigned short i = 0; i < NOV; i++){
			if (fa[i] != i) {
				prev2[p] = last2[i];
				last2[i] = p;
				other2[p] = fa[i];
				prev2[p+NOV-1] = last2[fa[i]];
				last2[fa[i]] = p+NOV-1;
				other2[p+NOV-1] = i;
			}
		}
	}
}
/*
void cudaSeperate()
{
	unsigned int *d_prev = NULL, *d_last = NULL; 
	unsigned short *d_other = NULL;
	float *d_weight = NULL;
	bool (*d_inlgroup1)[NOV] = NULL, (*d_inlgroup2)[NOV] = NULL;

	bool inlgroupp[NOV][NOV] = {false};

	bool *d_left = NULL;
	unsigned short *d_todeal = NULL;
	unsigned short todeal[NOV] = {0};
	short num_deal1 = -1;
	for (unsigned short i = 1; i < NOV; i++)
		if (weight1[i]>dslen1){
			num_deal1++;
			todeal[num_deal1] = i;
		}
	hipMalloc((void **)&d_todeal, sizeof(todeal));
	hipMemcpy(d_todeal, todeal, sizeof(todeal), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_prev, (2*NOV-1)*sizeof(unsigned int));	
	hipMemcpy(d_prev, prev1, (2*NOV-1)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_last, NOV*sizeof(unsigned int));
	hipMemcpy(d_last, last1, NOV*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_other, (2*NOV-1)*sizeof(unsigned short));
	hipMemcpy(d_other, other1, (2*NOV-1)*sizeof(unsigned short), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_inlgroup1, NOV*NOV*sizeof(bool));
	hipMalloc((void **)&d_weight, (2*NOV-1)*sizeof(float));
	hipMemcpy(d_weight, weight1, (2*NOV-1)*sizeof(float), hipMemcpyHostToDevice);
	int num_threads = (num_deal1+1)>1024?1024:(num_deal1+1);
	int num_blocks = (num_deal1+1) / num_threads+((num_deal1+1) % num_threads == 0?0:1);
	sep_mst<<<num_blocks,num_threads>>>(d_todeal, d_prev, d_last, d_other, d_weight, d_inlgroup1);	
	hipMemcpy(inlgroup, d_inlgroup1, NOV*NOV*sizeof(bool), hipMemcpyDeviceToHost);

	short num_deal2 = -1;
	for (unsigned short i = 1; i < NOV; i++)
		if (weight2[i]>dslen1){
			num_deal2++;
			todeal[num_deal2] = i;
		}
	hipMemcpy(d_todeal, todeal, sizeof(todeal), hipMemcpyHostToDevice);
	hipMemcpy(d_prev, prev2, (2*NOV-1)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_last, last2, NOV*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_other, other2, (2*NOV-1)*sizeof(unsigned short), hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight2, (2*NOV-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_inlgroup2, NOV*NOV*sizeof(bool));
	num_threads = (num_deal2+1)>1024?1024:(num_deal2+1);
	num_blocks = (num_deal2+1) / num_threads+((num_deal2+1) % num_threads == 0?0:1);
	sep_mst<<<num_blocks,num_threads>>>(d_todeal, d_prev, d_last, d_other, d_weight, d_inlgroup2);

	hipMemcpy(inlgroupp, d_inlgroup2, sizeof(inlgroupp), hipMemcpyDeviceToHost);

	hipFree(d_prev);
	hipFree(d_last);
	hipFree(d_other);
	hipFree(d_weight);
	hipMalloc((void **)&d_left, NOV*sizeof(bool));
	num_threads = (num_deal1+1)>1024?1024:(num_deal1+1);
	num_blocks = (num_deal1+1) / num_threads+((num_deal1+1) % num_threads == 0?0:1);
	reduct<<<num_blocks,num_threads>>>(d_inlgroup1, d_inlgroup2, d_left, epsilon, theta);
	hipMemcpy(inlgroup, d_inlgroup1, NOV*NOV*sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(left, d_left, NOV*sizeof(bool), hipMemcpyDeviceToHost);
	hipFree(d_left);
	hipFree(d_inlgroup1);
	hipFree(d_inlgroup2);

	unsigned short p = 1;
	FILE *fp;
	fp = fopen("e:\\data_out.txt", "w+");
	for (unsigned short i = 0; i <= num_deal1; i++)
		if (left[i]){
			fprintf(fp, "+++++++++++++++++++ %d +++++++++++++++++++\n", p);
			for (unsigned short j = 0; j < NOV; j++) {
				if (inlgroup[i][j]) {
					fprintf(fp, "%4d", j);
				}
			}
			fprintf(fp,"\n");
			for (unsigned short j = 0; j < NOV; j++){
				if (!inlgroup[i][j]) {
					fprintf(fp, "%4d", j);
				}
			}
			fprintf(fp,"\n");
			fprintf(fp,"------------------------------------------\n");
			p++;
		}
	fclose(fp);
}
*/
bool seperate(unsigned short *group, unsigned short rts)
{
	short t = -1;
	unsigned short *q;
	unsigned short head = 0, tail = 0;
	bool *v;
	float dslen1 = 0, dslen2 = 0;
	unsigned short pnum1 = 0, pnum2 = 0;
	bool etocut1[2 * NOV - 1] = {false}, etocut2[2 * NOV - 1] = {false};

	q = (unsigned short *)malloc(NOV*sizeof(short));
	v = (bool *)malloc((2*NOV-1)*sizeof(bool));
	memset(v,0,(2*NOV-1)*sizeof(bool));

	for (unsigned short i = 0; i < NOV; i++)
		if (group[i] == rts){
			t = i;
			break;
		}
	if (t == -1) return false;
	v[t] = true;
	q[tail] = t;
	while (head >= tail){
		unsigned int p;
		p = last1[q[tail]];
		while (p){
			if (!v[other1[p]] && group[other1[p]] == rts){
				v[other1[p]] = true;
				etocut1[p] = true;
				pnum1++;
				dslen1 += weight1[p];
				head++;
				q[head] = other1[p];
			}
			p = prev1[p];
		}
		tail++;
	}
	dslen1 /= pnum1;
	memset(v, 0, (2*NOV-1)*sizeof(bool));
	v[t] = true;
	head = tail = 0;
	while (head >= tail){
		unsigned short p;
		p = last2[q[tail]];
		while (p){
			if (!v[other2[p]] && group[other2[p]] == rts){
				v[other2[p]] = true;
				etocut2[p] = true;
				pnum2++;
				dslen2 += weight2[p];
				head++;
				q[head] = other2[p];
			}
			p = prev2[p];
		}
		tail++;
	}
	dslen2 /= pnum2;
	free(q);
	free(v);

	unsigned int *d_prev = NULL, *d_last = NULL; 
	unsigned short *d_other = NULL;
	float *d_weight = NULL;
	bool (*inlgroup1)[NOV];//, (*inlgroup2)[NOV];
	bool (*d_inlgroup1)[NOV] = NULL, (*d_inlgroup2)[NOV] = NULL;
	bool *d_left = NULL;
	unsigned short *d_todeal = NULL;
	unsigned short todeal1[NOV] = {0}, todeal2[NOV] = {0};
	short num_deal1 = -1;
	unsigned short *d_group = NULL;

	inlgroup1 = (bool (*) [NOV])malloc(NOV*NOV*sizeof(bool));
	//inlgroup2 = (bool (*) [NOV])malloc(NOV*NOV*sizeof(bool));

	for (unsigned short i = 1; i < NOV; i++)
		if (weight1[i]>dslen1 && etocut1[i]){
			num_deal1++;
			todeal1[num_deal1] = i;
		}
	if (num_deal1 < 0) return false;
	hipMalloc((void **)&d_todeal, sizeof(todeal1));
	hipMalloc((void **)&d_prev, (2*NOV-1)*sizeof(unsigned int));	
	hipMalloc((void **)&d_last, NOV*sizeof(unsigned int));
	hipMalloc((void **)&d_other, (2*NOV-1)*sizeof(unsigned short));
	hipMalloc((void **)&d_inlgroup1, NOV*NOV*sizeof(bool));
	hipMalloc((void **)&d_weight, (2*NOV-1)*sizeof(float));
	hipMalloc((void **)&d_group, NOV * sizeof(short));
	hipMemcpy(d_todeal, todeal1, sizeof(todeal1), hipMemcpyHostToDevice);
	hipMemcpy(d_prev, prev1, (2*NOV-1)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_last, last1, NOV*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_other, other1, (2*NOV-1)*sizeof(unsigned short), hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight1, (2*NOV-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_group, group, NOV * sizeof(short), hipMemcpyHostToDevice);
	int num_threads = (num_deal1+1)>1024?1024:(num_deal1+1);
	int num_blocks = (num_deal1+1) / num_threads+((num_deal1+1) % num_threads == 0?0:1);
	sep_mst2<<<num_blocks,num_threads>>>(d_todeal, d_prev, d_last, d_other, d_weight, d_inlgroup1, d_group, rts);	
	hipMemcpy(inlgroup1, d_inlgroup1, (num_deal1+1)*NOV*sizeof(bool), hipMemcpyDeviceToHost);
	
	short num_deal2 = -1;
	for (unsigned short i = 1; i < NOV; i++)
		if (weight2[i]>dslen2 && etocut2[i]){
			num_deal2++;
			todeal2[num_deal2] = i;
		}
	if (num_deal2 < 0) return false;
	hipMemcpy(d_todeal, todeal2, sizeof(todeal2), hipMemcpyHostToDevice);
	hipMemcpy(d_prev, prev2, (2*NOV-1)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_last, last2, NOV*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_other, other2, (2*NOV-1)*sizeof(unsigned short), hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight2, (2*NOV-1)*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_inlgroup2, NOV*NOV*sizeof(bool));
	num_threads = (num_deal2+1)>1024?1024:(num_deal2+1);
	num_blocks = (num_deal2+1) / num_threads+((num_deal2+1) % num_threads == 0?0:1);
	sep_mst2<<<num_blocks,num_threads>>>(d_todeal, d_prev, d_last, d_other, d_weight, d_inlgroup2, d_group, rts);
	//hipMemcpy(inlgroup2, d_inlgroup2, (num_deal2+1)*NOV*sizeof(bool), hipMemcpyDeviceToHost);
	
	hipFree(d_prev);
	hipFree(d_last);
	hipFree(d_other);
	hipFree(d_weight);

	hipMalloc((void **)&d_left, NOV*sizeof(bool));
	num_threads = (num_deal1+1)>1024?1024:(num_deal1+1);
	num_blocks = (num_deal1+1) / num_threads+((num_deal1+1) % num_threads == 0?0:1);
	reduct<<<num_blocks,num_threads>>>(d_inlgroup1, d_inlgroup2, d_left, epsilon, theta, pnum1);
	hipMemcpy(left, d_left, NOV*sizeof(bool), hipMemcpyDeviceToHost);
	hipFree(d_left);
	hipFree(d_inlgroup1);
	hipFree(d_inlgroup2);

	float max = 0;
	short fin = -1;
	for (unsigned short i = 0; i < num_deal1+1; i++)
		if (left[i] && weight1[todeal1[i]] > max){
			max = weight1[todeal1[i]];
			fin = i;
		}
	if (fin == -1) return false;
	rank++;

	for (unsigned short i = 0; i < NOV; i++)
		if (!inlgroup1[fin][i] && group[i] == rts)
			group[i] = rank;
	free(inlgroup1);
	return true;
	
}

void cudaSeperate2()
{
	unsigned short group[NOV] = {0};
	unsigned short q[2 * NOV] = {0};
	unsigned short head = 0, tail = 0;
	while (head >= tail){
		while (seperate(group, q[tail])){
			head++;
			q[head] = rank;
		}
		tail++;
	}

	FILE *fp;
	fp = fopen("e:\\data_out.txt", "w+");
	unsigned short p = 0, r = 1;
	bool v[NOV] = {false};
	while (p < NOV){
		unsigned short start = 0;
		fprintf(fp,"group %d:\n", r);
		while (v[start])
			start++;
		for (unsigned short i = start; i < NOV; i++)
			if (!v[i] && group[i] == group[start]){
				fprintf(fp,"%d\t\t", i);
				v[i] = true;
				p++;
			}
		fprintf(fp,"\n");
		r++;
	}
	fclose(fp);
}

int main()
{
	unsigned int *v;
	unsigned short *e;
	float *w;

	prev = (unsigned int *)malloc((2 * NOE + 2)*sizeof(unsigned int));
	last = (unsigned int *)malloc(NOV*sizeof(int));
	other = (unsigned short *)malloc((2 * NOE + 2)*sizeof(unsigned short));
	a = (unsigned char (*)[NOV])malloc(NOV*NOV*sizeof(unsigned char));
	memset(prev,0,(2 * NOE + 2)*sizeof(unsigned int));
	memset(last,0,NOV*sizeof(int));
	memset(other,0,(2 * NOE + 2)*sizeof(unsigned short));
	memset(a,0,NOV*NOV*sizeof(unsigned char));
	//readfile_edgelist();
	readfile_matrix();

	printf("%u\n",m);
	
	cudaMatrixMul();
	fa = (unsigned short *)malloc(NOV*sizeof(unsigned short));
	isleaf = (bool *)malloc(NOV*sizeof(bool));
	memset(fa,0,NOV*sizeof(unsigned short));
	memset(isleaf,0,NOV*sizeof(bool));

	search_leaves();

	d = (float (*)[NOV])malloc(NOV*NOV*sizeof(float));
	memset(d, 0, NOV*NOV*sizeof(float));

	cudaComputeDistanceMatrix();
	
	free(prev);
	free(last);
	free(other);
	free(a);

	v = (unsigned int *)malloc((NOV+1)*sizeof(unsigned int));
	e = (unsigned short *)malloc(NOV*NOV*sizeof(unsigned short));
	w = (float *)malloc(NOV*NOV*sizeof(float));
	memset(v,0,(NOV+1)*sizeof(unsigned int));
	memset(e,0,NOV*NOV*sizeof(unsigned short));
	memset(w,0,NOV*NOV*sizeof(float));

	unsigned int p, i;
	for (p = i = 0; i < n; i++){
		if (!isleaf[i]){
			v[i] = p;
			for (unsigned short j = 0; j < n; j++){
				if (!isleaf[j] && i!=j && d[i][j] < 998){
					e[p]=j;
					w[p]=d[i][j];
					p++;
				}
			}
		}
	}
	v[NOV - num_leaves] = p;
	free(d);

	prev1 = (unsigned int *)malloc((2*NOV-1)*sizeof(unsigned int));
	prev2 = (unsigned int *)malloc((2*NOV-1)*sizeof(unsigned int));
	last1 = (unsigned int *)malloc(NOV*sizeof(unsigned int));
	last2 = (unsigned int *)malloc(NOV*sizeof(unsigned int));
	other1 = (unsigned short *)malloc((2*NOV-1)*sizeof(unsigned short));
	other2 = (unsigned short *)malloc((2*NOV-1)*sizeof(unsigned short));
	weight1 = (float *)malloc((2*NOV-1)*sizeof(float));
	weight2 = (float *)malloc((2*NOV-1)*sizeof(float));
	memset(prev1,0,(2*NOV-1)*sizeof(unsigned int));
	memset(last1,0,NOV*sizeof(unsigned int));
	memset(other1,0,(2*NOV-1)*sizeof(unsigned short));
	memset(weight1,0,(2*NOV-1)*sizeof(float));
	memset(prev2,0,(2*NOV-1)*sizeof(unsigned int));
	memset(last2,0,NOV*sizeof(unsigned int));
	memset(other2,0,(2*NOV-1)*sizeof(unsigned short));
	memset(weight2,0,(2*NOV-1)*sizeof(float));

	prim_mst(v, e, w, 1);
	prim_mst(v, e, w, 2);
	free(fa);
	free(isleaf);

	dslen1 /= (NOV-1-num_leaves);
	dslen2 /= (NOV-1-num_leaves);

	inlgroup = (bool (*)[NOV])malloc(NOV*NOV*sizeof(bool));
	left = (bool *)malloc(NOV*sizeof(bool));
	memset(inlgroup,0,NOV*NOV*sizeof(bool));
	memset(left,0,NOV*sizeof(bool));

	//cudaSeperate();
	cudaSeperate2();

	free(left);
	free(inlgroup);
	free(prev1);
	free(prev2);
	free(last1);
	free(last2);
	free(other1);
	free(other2);
	free(weight1);
	free(weight2);
	
	return 0;
}

