#include "hip/hip_runtime.h"
#include "abschlussarbeit.cuh"

#define left left2d[tid]
#define right right2d[tid]

__device__ void findexroute(unsigned int tid,
							const unsigned short tk,
							const unsigned int *prev,
							const unsigned int *last,
							const unsigned short *other,
							bool *r,
							unsigned short *match,
							bool *res,
							unsigned int *stackp,
							unsigned short *stackk,
							bool *stackf,
							bool *visit)
{
	unsigned short k = tk;
	volatile unsigned int p = last[k];
	unsigned short sp = 0;
	
	for (unsigned int i = 0; i < NOV; i++){
		visit[i] = false;
		stackf[i] = stackk[i] = stackp[i] = 0;
	}

	while (p) {
		if (stackf[sp]) {
			match[other[p]] = k;
			if (sp) {
				sp--;
				p = stackp[sp];
				k = stackk[sp];
				stackf[sp] = true;
			}else{
				*res = true;
				return;
			}
		}else {
			if (r[other[p]] && !visit[other[p]]) {
				visit[other[p]] = true;
				if (match[other[p]] == 65535) {
					if (!sp) {
						match[other[p]] = k;
						*res = true;
						return;
					}else{
						sp--;
						match[other[p]] = k;
						p = stackp[sp];
						k = stackk[sp];
						stackf[sp] = true;
					}
				}else{
					stackp[sp] = p;
					stackk[sp] = k;
					sp++;
					k = match[other[p]];
					p = last[k];
				}
			}else{
				p = prev[p];
			}
		}
	}
	*res = false;
	return;
}

__global__ void computeDistanceMatrix(const unsigned int *prev, const unsigned int *last,
									  const unsigned short *other, const bool *isleaf,
									  float (*d)[NOV], float delta, 
									  //unsigned int (*stackp2d)[NOV], unsigned short (*stackk2d)[NOV], bool (*stackf2d)[NOV], bool (*visit2d)[NOV], 
									  bool (*left2d)[NOV], bool (*right2d)[NOV], unsigned char (*a)[NOV])
{
	unsigned short tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < NOV && !isleaf[tid]){
		for (unsigned short i = tid + 1; i < NOV; i++) if (!isleaf[i] && a[tid][i]<4) {
			for (unsigned int j = 0; j < NOV; j++)
				left[j] = right[j] = 0;
			unsigned short num_left = 0;
			unsigned short num_right = 0;
			unsigned int ps = last[tid], pt = last[i];
			unsigned short t = 1;
			while (ps) {
				unsigned short otherps = other[ps];
				if (otherps != i){
					left[otherps] = true;
					num_left++;
				}
				ps = prev[ps];
			}
			while (pt) {
				unsigned short otherpt = other[pt];
				if (otherpt == tid) {
					d[tid][i]+=1;
					pt = prev[pt];
					continue;
				}
				if (left[otherpt]) {
					d[tid][i]+=0.5;
					left[otherpt] = false;
					num_left--;
					pt = prev[pt];
					continue;
				}
				unsigned int p = last[otherpt];
				bool isright = false;
				while (p) {
					unsigned short otherp = other[p];
					if (left[otherp]) {
						isright = true;
						t++;
					}
					p = prev[p];
				}
				if (isright) {
					num_right++;
					right[otherpt] = true;
				}
				pt = prev[pt];
			}
			if (num_left != 0 && num_right != 0){
				if (num_left == 1 || num_right == 1) {
					d[tid][i] += 0.3333333;
				}else{
					unsigned int ans = 0;
					if (num_left<num_right)
						ans = num_left;
					else
						ans = num_right;
					d[tid][i] += ans * 0.3333333;
				}
			}
		}
		for (unsigned int i = tid+1; i < NOV; i++)
			d[i][tid] = d[tid][i];
		__syncthreads();
		for (unsigned int i = 0; i < NOV; i++)
			if (i != tid){
				if (d[tid][i]<0.00001 && d[tid][i]>-0.00001)
					d[tid][i] = 9999;
				else
					if (a[tid][i])
						d[tid][i] = (1/d[tid][i])*(1-delta);
					else
						d[tid][i] = 1/d[tid][i];
			}
	}
}

__global__ void matrixMulCUDA(unsigned char (*C)[NOV], 
							  unsigned char (*A)[NOV], 
							  unsigned char (*B)[NOV])
{
	unsigned int tid = blockDim.x * blockIdx.x +threadIdx.x;
	unsigned int cx = tid / NOV;
	unsigned int cy = tid % NOV;
	if (cx<NOV && cy<NOV){
		C[cx][cy] = 0;
#pragma unroll
		for (unsigned int i = 0; i < NOV; i++)
			C[cx][cy]|=A[cx][i]&B[i][cy];
	}
}

__global__ void matrixfix(unsigned char (*A)[NOV], 
						  const unsigned char (*B)[NOV],
						  const unsigned char (*C)[NOV],
						  const unsigned char (*D)[NOV])
{
	unsigned int tid = blockDim.x * blockIdx.x +threadIdx.x;
	unsigned int cx = tid / NOV;
	unsigned int cy = tid % NOV;
	if (cx >= NOV || cy >= NOV) return;
	if (cx == cy){
		A[cx][cy] = 0;
		return;
	}
	if (B[cx][cy]) {
		A[cx][cy] = 1;
		return;
	}
	if (C[cx][cy]) {
		A[cx][cy] = 2;
		return;
	}
	if (D[cx][cy]) {
		A[cx][cy] = 3;
		return;
	}
	A[cx][cy] = 99;
}

