#include "hip/hip_runtime.h"
#include "min_reduction.cuh"

__device__ void warpReduce1(volatile float *sdata, volatile unsigned int *sid, float *ssweight, unsigned int *ssid,
							unsigned int tid, unsigned int blockSize, unsigned int n) 
{
	if (blockSize >= 64) if (sdata[tid] > sdata[tid + 32] && tid+32<n) {
		*ssweight = sdata[tid] = sdata[tid + 32];
		*ssid = sid[tid] = sid[tid+32];
	}
	if (blockSize >= 32) if (sdata[tid] > sdata[tid + 16] && tid+16<n) {
		*ssweight = sdata[tid] = sdata[tid + 16];
		*ssid = sid[tid]= sid[tid+16];
	}

	if (blockSize >= 16) if (sdata[tid] > sdata[tid + 8] && tid+8<n)  {
		*ssweight = sdata[tid] = sdata[tid + 8];
		*ssid = sid[tid] = sid[tid+8];
	}
	if (blockSize >= 8)  if (sdata[tid] > sdata[tid + 4] && tid+4<n)  {
		*ssweight = sdata[tid] = sdata[tid + 4];
		*ssid = sid[tid] = sid[tid+4];
	}
	if (blockSize >= 4)  if (sdata[tid] > sdata[tid + 2] && tid+2<n)  {
		*ssweight = sdata[tid] = sdata[tid + 2];
		*ssid = sid[tid]= sid[tid+2];
	}
	if (blockSize >= 2)  if (sdata[tid] > sdata[tid + 1] && tid+1<n)  {
		*ssweight = sdata[tid] = sdata[tid + 1];
		*ssid = sid[tid] = sid[tid+1];
	}
}

__device__ void warpReduce2(volatile float *sdata, volatile unsigned int *sid, float *ssweight, unsigned int *ssid,
							unsigned int tid, unsigned int blockSize, unsigned int n) 
{
	if (blockSize >= 64) if (sdata[tid] > sdata[tid + 32] && tid+32<n) {
		*ssweight = sdata[tid] = sdata[tid + 32];
		*ssid = sid[tid] = sid[tid+32];
	}
	if (blockSize >= 32) if (sdata[tid] > sdata[tid + 16] && tid+16<n) {
		*ssweight = sdata[tid] = sdata[tid + 16];
		*ssid = sid[tid]= sid[tid+16];
	}

	if (blockSize >= 16) if (sdata[tid] > sdata[tid + 8] && tid+8<n)  {
		*ssweight = sdata[tid] = sdata[tid + 8];
		*ssid = sid[tid] = sid[tid+8];
	}
	if (blockSize >= 8)  if (sdata[tid] > sdata[tid + 4] && tid+4<n)  {
		*ssweight = sdata[tid] = sdata[tid + 4];
		*ssid = sid[tid] = sid[tid+4];
	}
	if (blockSize >= 4)  if (sdata[tid] > sdata[tid + 2] && tid+2<n)  {
		*ssweight = sdata[tid] = sdata[tid + 2];
		*ssid = sid[tid]= sid[tid+2];
	}
	if (blockSize >= 2)  if (sdata[tid] > sdata[tid + 1] && tid+1<n)  {
		*ssweight = sdata[tid] = sdata[tid + 1];
		*ssid = sid[tid] = sid[tid+1];
	}
}

__global__ void min_reduction1(float *d_iweight, float *d_oweight,
							  unsigned int *d_oid, unsigned int n) 
{
	__shared__ float sdata[1024];
	__shared__ unsigned int sid[1024];
	unsigned int tid = threadIdx.x;
	unsigned int blockSize = blockDim.x;
	unsigned int i = blockIdx.x * blockSize + tid;
	unsigned int gridSize = gridDim.x * blockDim.x;
	float ssweight;
	unsigned int ssid;
	sdata[tid] = 0;
	sid[tid] = 0;
	if ((i < gridSize) && (i >= n))
		ssweight = sdata[tid] = MAX_WEIGHT;
	if (i < n){
		ssweight = sdata[tid] = d_iweight[i];
		ssid = sid[i] = i;
		if (sdata[tid] == 0)
			ssweight = sdata[tid] = MAX_WEIGHT;
	}
	__syncthreads();
	if (blockSize == 1024) {
		if ((tid < 512) && (sdata[tid] > sdata[tid + 512]) && tid+512<n){
			ssweight = sdata[tid] = sdata[tid+512];
			ssid = sid[tid] = sid[tid+512];
		}
		__syncthreads();
	}
	if (blockSize >= 512) {
		if ((tid < 256) && (sdata[tid] > sdata[tid + 256]) && tid+256<n) {
			ssweight = sdata[tid] = sdata[tid + 256];
			ssid = sid[tid] = sid[tid+ 256];
		}
		__syncthreads(); 
	}
	if (blockSize >= 256) { 
		if ((tid < 128) && (sdata[tid] > sdata[tid + 128]) && tid+128<n) {
			ssweight = sdata[tid] = sdata[tid + 128]; 
			ssid = sid[tid] = sid[tid+128];
		}
		__syncthreads(); 
	}
	if (blockSize >= 128) { 
		if ((tid < 64) &&  (sdata[tid] > sdata[tid + 64]) && tid+64<n){
			ssweight = sdata[tid] = sdata[tid + 64]; 
			ssid = sid[tid] = sid[tid + 64];
		}
		__syncthreads(); 
	}
	if (tid < 32) warpReduce1(sdata, sid, &ssweight, &ssid, tid, blockSize, n);
	if (tid == 0) {
		*d_oid = sid[0];
		*d_oweight = sdata[0];
	}
}

__global__ void min_reduction2(float *d_iweight, float *d_oweight,
							  unsigned int *d_oid, unsigned int *d_iid,
							  unsigned int n) 
{
	__shared__ float sdata[5];
	__shared__ unsigned int sid[5];
	unsigned int tid = threadIdx.x;
	unsigned int blockSize = blockDim.x;
	unsigned int i = blockIdx.x * blockSize + tid;
	unsigned int gridSize = gridDim.x * blockDim.x;

	sdata[tid] = d_iweight[i];
	sid[i] = d_iid[i];
	if (sdata[tid] == 0)
		sdata[tid] = MAX_WEIGHT;

	if (blockSize >= 4)  if (sdata[tid] > sdata[tid + 2] && tid+2<n)  {
		sdata[tid] = sdata[tid + 2];
		sid[tid]= sid[tid+2];
	}
	if (blockSize >= 2)  if (sdata[tid] > sdata[tid + 1] && tid+1<n)  {
		sdata[tid] = sdata[tid + 1];
		sid[tid] = sid[tid+1];
	}
	*d_oid= sid[0];
	*d_oweight= sdata[0];
}
