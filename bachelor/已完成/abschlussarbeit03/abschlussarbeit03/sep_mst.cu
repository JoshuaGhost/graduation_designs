#include "hip/hip_runtime.h"
#include "sep_mst.cuh"

__global__ void sep_mst(unsigned short *todeal,
						unsigned int *prev, unsigned int *last, unsigned short *other, 
						float *weight, bool (*inlgroup)[NOV])
{
	bool l[NOV] = {false};
	unsigned short tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned short td = todeal[tid];
	unsigned short q[NOV] = {0};
	unsigned short head = 0;
	unsigned short tail = 0;
	l[0] = true;
	while (head >= tail){
		unsigned int p = last[q[tail]];
		while (p){
			if (p != td && p != td + NOV - 1){
				unsigned short otherp = other[p];
				if (!l[otherp]){
					head++;
					q[head] = otherp;
					l[otherp] = true;
				}
			}
			p = prev[p];
		}
		tail++;
	}
	for (unsigned i = 0; i < NOV; i++)
		inlgroup[tid][i] = l[i];
}

__global__ void reduct(bool (*inlgroup1)[NOV], bool (*inlgroup2)[NOV], bool *left, 
					   unsigned short epsilon, unsigned short theta, unsigned short pnum1)
{
	unsigned int tid = blockDim.x *blockIdx.x + threadIdx.x;
	left[tid] = false;
	bool l1[NOV] = {false};
	unsigned short sum = 0;
	for (unsigned short i = 0; i < NOV; i++){
		if (inlgroup1[tid][i]){
			l1[i] = true;
			sum++;
		}
	}
	if (sum >= epsilon && pnum1-sum>=epsilon){
		for (unsigned short i = 0; i < NOV; i++){
			unsigned short t = 0;
			while (t < NOV && !inlgroup2[i][t]) t++;
			if (t == NOV) {
				left[tid] = false;
				continue;
			} else {
				bool legal = true;
				unsigned char dismatch = 0;
				for (unsigned short j = t; j < NOV; j++){
					if (l1[j] ^ inlgroup2[i][j]){
						dismatch++;
					}
				}
				if (dismatch < theta || pnum1 - dismatch < theta){
					left[tid] = true;
					break;
				}
			}
		}
	} else {
		left[tid] = false;
	}
}

__global__ void sep_mst2(unsigned short *todeal,
						unsigned int *prev, unsigned int *last, unsigned short *other, 
						float *weight, bool (*inlgroup)[NOV], unsigned short *group, unsigned short rts)
{
	bool l[NOV] = {false};
	unsigned short tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned short td = todeal[tid];
	unsigned short q[NOV] = {0};
	unsigned short head = 0;
	unsigned short tail = 0;
	for (unsigned short i = 0; i < NOV; i++)
		if (group[i] == rts){
			l[i] = true;
			q[tail] = i;
			break;
		}
	while (head >= tail){
		unsigned int p = last[q[tail]];
		while (p){
			if (p != td && p != td + NOV - 1){
				unsigned short otherp = other[p];
				if (!l[otherp] && group[otherp] == rts){
					head++;
					q[head] = otherp;
					l[otherp] = true;
				}
			}
			p = prev[p];
		}
		tail++;
	}
	for (unsigned i = 0; i < NOV; i++)
		inlgroup[tid][i] = l[i];
}